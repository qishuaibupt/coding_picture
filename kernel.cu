#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "math.h"
#include <stdio.h>

__global__ void computeOut(unsigned char *g_indata, unsigned char *g_odata, int imgw, int imgh, int outw, int outh, int viewnum, float LineNum, float InclinationAngle, float MoveValue, bool ifReverse)
{
	int y = blockDim.y * blockIdx.y + threadIdx.y;//�����߳�ID
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	

	if (y<outh && x<outw)
	{
		int i = 0;//��Ӧ�Ӳ�ͼ���

		float step_value = 1.0 / viewnum * LineNum;
		for (int k = 0; k < 3; k++)
		{
			float value_pixel = x * 3 + 3 * y * InclinationAngle + 2 - k;//bgr->rgb��
			//float value_pixel = x * 3 + 3 * y * InclinationAngle ;//bgr->rgb��
			float judge_value = value_pixel - int(value_pixel / LineNum) * LineNum;
			if (0 > judge_value)
			{
				judge_value = judge_value + LineNum;
			}
			int view_point_number = floor(judge_value / step_value);//��0��ʼ �����Ӧ��vied_point


			if (MoveValue < 0)//λ��
				MoveValue = viewnum - abs(MoveValue);
			view_point_number = (view_point_number + (int)MoveValue) % viewnum;


			if (ifReverse)//default true ͨ��vied_point�����Ӳ�ͼ���
			{
				i = viewnum - view_point_number - 1;//��ת
			}
			else
			{
				i = view_point_number;
			}
			unsigned char* temp = g_indata;
			temp = temp + i*imgw *imgh * 3;//��ͼ�񲢲������Դ洢


			if (imgw == outw&& imgh == outh)//��out��ĳһ���ص�����Ӧ���ӵ�
			{//����ߴ���ͬ
				g_odata[y*outw * 4 + 4 * x + k] = temp[y*imgw * 3 + 3 * x + 2 - k]; 
			}
			else{//����ߴ粻��ͬ	
				float orgx = x * imgw / outw;
				float orgy = y * imgh / outh;
				float u = ceil(orgx) - orgx;
				float v = ceil(orgy) - orgy;
				int orgx_i = ceil(orgx);
				int orgy_i = ceil(orgy);
				//˫���Բ�ֵ  
				int tempL = temp[orgy_i * imgw * 3 + 3 * orgx_i + k] * u * v +
					temp[(orgy_i + 1) * imgw * 3 + 3 * orgx_i + k] * (1 - v) * u +
					temp[orgy_i * imgw * 3 + 3 * (orgx_i + 1) + k] * v* (1 - u) +
					temp[(orgy_i + 1) * imgw * 3 + 3 * (orgx_i + 1) + k] * (1 - v) * (1 - u);
				g_odata[y*outw * 4 + 4 * x + 2 - k] = tempL;
				//g_odata[y*outw * 4 + 4 * x + k] = tempL;
			}	
		}//RGB��������
	}//ID��������
}

extern "C"
{
	void launch_cudaProcess_code(unsigned char* g_inframe, unsigned char* g_odata, int in_w, int in_h, int out_w, int out_h, int viewnum, bool ifReverse, int LineNum, float InclinationAngle, int MoveValue)
	{
		hipError_t error = hipSuccess;

		dim3 thread = dim3(16, 16);   // block dimensions are fixed to be 256 threads
		dim3 block = dim3((out_w + thread.x - 1) / thread.x, (out_h + thread.y - 1) / thread.y);

		hipDeviceSynchronize(); //�����豸�ȴ�
		computeOut << <block, thread >> >(g_inframe, g_odata, in_w, in_h, out_w, out_h, viewnum, LineNum, InclinationAngle, MoveValue, ifReverse);

		error = hipGetLastError();
		if (error != hipSuccess)
		{
			printf("launch_cudaProcess() failed to launch error = %d\n", error);
		}
	}
}

